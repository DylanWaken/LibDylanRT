#include "hip/hip_runtime.h"
//
// Created by dylan on 1/25/23.
//

#include "PhongShading.cuh"
#include "UtilFunctions.cuh"
#include "chrono"

#define AABB_PREFETCHED_LAYER_SIZE (1+ 2 + 4 + 8 + 16 + 32 + 64 + 128 + 256 + 512)
#define AABB_PREFETCHED_LAYERS 11
#define AABB_SEARCH_STACK_DEPTH 24

namespace dylanrt {

    struct solvedParams{
        bool solved;
        float3 trigParams;
        float3 trigV1;
        float3 trigV2;
        float3 trigV3;
        unsigned int trigIndex;
    };

    //we use slab method to check if a ray intersects with a box
    //basically we treat the box as pairs of parellel planes and gradually clip the ray
    //if the ray is still alive after clipping, then it intersects with the box
    __device__ __forceinline__ bool rayBoxIntersect(float3 max, float3 min, float3 e, float3 inverseD){
        float t1x = (min.x - e.x) * inverseD.x;
        float t2x = (max.x - e.x) * inverseD.x;

        float t1y = (min.y - e.y) * inverseD.y;
        float t2y = (max.y - e.y) * inverseD.y;

        float t1z = (min.z - e.z) * inverseD.z;
        float t2z = (max.z - e.z) * inverseD.z;

        float tmin = fmaxf(fmaxf(fminf(t1x, t2x), fminf(t1y, t2y)), fminf(t1z, t2z));
        float tmax = fminf(fminf(fmaxf(t1x, t2x), fmaxf(t1y, t2y)), fmaxf(t1z, t2z));

        if(tmax < 0) return false;
        return tmax >= tmin;
    }

    __device__ __forceinline__ bool rayBoxIntersect(float maxX, float maxY, float maxZ
                                                 , float minX, float minY, float minZ
                                                 , float eX, float eY, float eZ,
                                                 float invDX, float invDY, float invDZ){
        float t1x = (minX - eX) * invDX;
        float t2x = (maxX - eX) * invDX;

        float t1y = (minY - eY) * invDY;
        float t2y = (maxY - eY) * invDY;

        float t1z = (minZ - eZ) * invDZ;
        float t2z = (maxZ - eZ) * invDZ;

        float tmin = fmaxf(fmaxf(fminf(t1x, t2x), fminf(t1y, t2y)), fminf(t1z, t2z));
        float tmax = fminf(fminf(fmaxf(t1x, t2x), fmaxf(t1y, t2y)), fmaxf(t1z, t2z));

        if(tmax < 0) return false;
        return tmax >= tmin;
    }

    __device__ __forceinline__ solvedParams findIntersect(float3 e, float3 d, float3 invD, AABBnode* nodeShared, AABBnode* nodes,
                                                          triangle* trigs, float3* vertices){
        //register of node indices:
        unsigned int nodeIndices[AABB_SEARCH_STACK_DEPTH];
        int nodeIndicesTop = 0;
        //push root node (since the zero node stands for root)
        nodeIndices[0] = 0;
        nodeIndicesTop++;

        solvedParams result = {false, {0, 0, 1e20}, {0, 0, 0},
                               {0, 0, 0}, {0, 0, 0}, 0};
        #pragma unroll
        while(nodeIndicesTop > 0){
            //read the node correspond to the current stack top
            unsigned int nodeIndex = nodeIndices[nodeIndicesTop - 1];
            nodeIndicesTop--;

            //raea the node correspondent to the current stack top
            AABBnode node = nodeIndex < AABB_PREFETCHED_LAYER_SIZE ? nodeShared[nodeIndex] : nodes[nodeIndex];

            //if is leaf, return the triangle index
            if(node.isLeaf){
                float3 v1 = vertices[trigs[node.trigIndex].indices.x];
                float3 v2 = vertices[trigs[node.trigIndex].indices.y];
                float3 v3 = vertices[trigs[node.trigIndex].indices.z];
                float3 trigParams = cramerIntersectSolver(e, d, v1, v2, v3);
                if (trigInRange(trigParams.x, trigParams.y)){
                    if(abs(trigParams.z) <= abs(result.trigParams.z)){
                        result.solved = true;
                        result.trigParams = trigParams;
                        result.trigV1 = v1;
                        result.trigV2 = v2;
                        result.trigV3 = v3;
                        result.trigIndex = node.trigIndex;
                    }
                }
                continue;
            }

            //find the overlap between the left and right nodes of the current node
            AABBnode left = node.left < AABB_PREFETCHED_LAYER_SIZE ? nodeShared[node.left] : nodes[node.left];
            AABBnode right = node.right < AABB_PREFETCHED_LAYER_SIZE ? nodeShared[node.right] : nodes[node.right];

            bool rayIntersectLeft = rayBoxIntersect(left.maxPoint, left.minPoint, e, invD);
            bool rayIntersectRight = rayBoxIntersect(right.maxPoint, right.minPoint, e, invD);
            if(!rayIntersectLeft && !rayIntersectRight){
                continue;
            }

            //if the ray intersects with the left node, push it to the stack
            //put the closer node on the top of the stack to reduce repetitive search
            if(rayIntersectLeft){
                nodeIndices[nodeIndicesTop] = node.left;
                nodeIndicesTop++;
            }
            if(rayIntersectRight){
                nodeIndices[nodeIndicesTop] = node.right;
                nodeIndicesTop++;
            }
        }

        //asm("trap;");
        return result;
    }

    template<const int BM>
    __launch_bounds__(BM, 2)
    __global__ void phongShadingDepthD(material* materials, triangle* trigs, float3* vertices, AABBnode* nodes, pointLight* lights,
                                       unsigned int numNodes, unsigned int numLights, CameraFrame* cameraFrame, float* imagePlane,
                                       unsigned int numPixls, float3 ambientLight){

        unsigned int tid = threadIdx.x;
        unsigned int bid = blockIdx.x;

        unsigned int begIndex = numPixls / gridDim.x * bid;
        unsigned int endIndex = gridDim.x == blockIdx.x + 1 ? numPixls : numPixls / gridDim.x * (bid + 1);

        //shared memory for the first layers:
        __shared__ AABBnode nodeShared[AABB_PREFETCHED_LAYER_SIZE];

        //read the camera frame
        CameraFrame frame = *cameraFrame;

        //copy range:
        unsigned int prefetchRange = min(numNodes, AABB_PREFETCHED_LAYER_SIZE);

        //all threads in the block prefetch the top layers
        //since these layers are used the most, we cache them in shared memory to improve the performance
        #pragma unroll
        for(auto i = tid; i < prefetchRange; i += BM){
            nodeShared[i] = nodes[i];
        }
        __syncthreads();

        //iterate through all pixels
        #pragma unroll
        for(auto i = begIndex + tid; i < endIndex; i += BM){
            //calculate pixel location
            float x = i%frame.resolutionX;
            float y = i/frame.resolutionX;

            //calculate the direction of the ray
            float picX = frame.imgTopLeft.x + (x) * ((frame.imgTopRight.x - frame.imgTopLeft.x)/frame.resolutionX) + (y) * ((frame.imgBottomLeft.x - frame.imgTopLeft.x)/frame.resolutionY);
            float picY = frame.imgTopLeft.y + (x) * ((frame.imgTopRight.y - frame.imgTopLeft.y)/frame.resolutionX) + (y) * ((frame.imgBottomLeft.y - frame.imgTopLeft.y)/frame.resolutionY);
            float picZ = frame.imgTopLeft.z + (x) * ((frame.imgTopRight.z - frame.imgTopLeft.z)/frame.resolutionX) + (y) * ((frame.imgBottomLeft.z - frame.imgTopLeft.z)/frame.resolutionY);

            //solve for ray direction
            float3 d = normalize3d(subtract3d(make_float3(picX, picY, picZ),frame.positionE));
            float3 invD = make_float3(1.0f/d.x, 1.0f/d.y, 1.0f/d.z);

            //find the closest triangle
            solvedParams trigParams = findIntersect(frame.positionE, d, invD, nodeShared, nodes, trigs, vertices);

            float z0 = abs(norm3d(subtract3d(frame.positionE, make_float3(0,0,0))));

            if (trigParams.solved){
                imagePlane[i] = z0 - abs(trigParams.trigParams.z);
                imagePlane[numPixls*1 + i] = z0 - abs(trigParams.trigParams.z);
                imagePlane[numPixls*2 + i] = z0 - abs(trigParams.trigParams.z);
            }else{
                imagePlane[i] = 0.0f;
                imagePlane[numPixls*1 + i] = 0.0f;
                imagePlane[numPixls*2 + i] = 0.0f;
            }
        }
    }

    template<const int PIXEL_PER_BLOCK>
    __global__ void phongShadingParaD(material* materials, triangle* trigs, float3* vertices, AABBnode* nodes, pointLight* lights,
                                      unsigned int numNodes, unsigned int numLights, CameraFrame* cameraFrame, float* imagePlane,
                                      unsigned int numPixls, float3 ambientLight){

        unsigned const int tid = threadIdx.x;
        unsigned const int bid = blockIdx.x;

        unsigned const int warpId = tid / 32;
        unsigned const int laneId = tid % 32;

        unsigned const int begIndex = numPixls / gridDim.x * bid;
        unsigned const int endIndex = gridDim.x == blockIdx.x + 1 ? numPixls : numPixls / gridDim.x * (bid + 1);

        unsigned const int BUFFER_SIZE = AABB_SEARCH_STACK_DEPTH * PIXEL_PER_BLOCK;

        __shared__ float directions[PIXEL_PER_BLOCK][3];
        __shared__ float invDirections[PIXEL_PER_BLOCK][3];
        __shared__ float solved[PIXEL_PER_BLOCK][3];

        //[pixel_ID ... | Node_access_request ...]
        extern __shared__ unsigned int stackData[];

        //shared memory for image plane, we can save registers by storing the image plane in shared memory
        __shared__ float imgTopLeft[3];
        __shared__ float imgTopRight[3];
        __shared__ float imgBottomLeft[3];
        __shared__ float eyePos[3];
        __shared__ unsigned int resolution[2];
        __shared__ int stackPtrPending[1];
        __shared__ int stackPtrProc[1];

        //read the camera frame
        if (tid == 0){
            imgTopRight[0] = cameraFrame->imgTopRight.x;
            imgTopRight[1] = cameraFrame->imgTopRight.y;
            imgTopRight[2] = cameraFrame->imgTopRight.z;

            imgTopLeft[0] = cameraFrame->imgTopLeft.x;
            imgTopLeft[1] = cameraFrame->imgTopLeft.y;
            imgTopLeft[2] = cameraFrame->imgTopLeft.z;

            imgBottomLeft[0] = cameraFrame->imgBottomLeft.x;
            imgBottomLeft[1] = cameraFrame->imgBottomLeft.y;
            imgBottomLeft[2] = cameraFrame->imgBottomLeft.z;

            eyePos[0] = cameraFrame->positionE.x;
            eyePos[1] = cameraFrame->positionE.y;
            eyePos[2] = cameraFrame->positionE.z;

            resolution[0] = cameraFrame->resolutionX;
            resolution[1] = cameraFrame->resolutionY;
        }
        __syncthreads();

        //start main loop
        #pragma unroll
        for (auto i = begIndex; i < endIndex; i+=PIXEL_PER_BLOCK){
            //precompute direction and invDirection
            if(tid < PIXEL_PER_BLOCK){
                //calculate pixel location
                float x = (i + tid)%resolution[0];
                float y = (i + tid)/resolution[0];

                //calculate the direction of the ray
                float picX = imgTopLeft[0] + (x) * ((imgTopRight[0] - imgTopLeft[0])/resolution[0]) + (y) * ((imgBottomLeft[0] - imgTopLeft[0])/resolution[1]);
                float picY = imgTopLeft[1] + (x) * ((imgTopRight[1] - imgTopLeft[1])/resolution[0]) + (y) * ((imgBottomLeft[1] - imgTopLeft[1])/resolution[1]);
                float picZ = imgTopLeft[2] + (x) * ((imgTopRight[2] - imgTopLeft[2])/resolution[0]) + (y) * ((imgBottomLeft[2] - imgTopLeft[2])/resolution[1]);

                //solve for ray direction
                float dX = picX - eyePos[0];
                float dY = picY - eyePos[1];
                float dZ = picZ - eyePos[2];

                directions[tid][0] = dX;
                directions[tid][1] = dY;
                directions[tid][2] = dZ;

                invDirections[tid][0] = 1.0f/dX;
                invDirections[tid][1] = 1.0f/dY;
                invDirections[tid][2] = 1.0f/dZ;

                solved[tid][0] = 0.0f;
                solved[tid][1] = 0.0f;
                solved[tid][2] = 0.0f;

                //assign pixel requests
                stackData[tid] = tid;
                //assign requests for the root node
                stackData[PIXEL_PER_BLOCK * AABB_SEARCH_STACK_DEPTH + tid] = 0;
                if(tid == 0) {
                    stackPtrPending[0] = PIXEL_PER_BLOCK;
                    stackPtrProc[0] = 0;
                }
            }

            __syncthreads();

            //start the first compute iteration
            int queued = 0;
            #pragma unroll
            while((queued = stackPtrPending[0] - stackPtrProc[0]) > 0){

                int readIndex = 0;
                //warp operation:
                if(laneId == 0){
                    readIndex = atomicAdd(stackPtrProc, min(queued, WARP_SIZE)) - min(queued, WARP_SIZE);
                }
                //get read index
                readIndex = __shfl_sync(0xffffffff, readIndex, 0) + (int)laneId;

                if(tid < queued) {

                    //compute the memory access index of this thread
                    readIndex = tid < queued ? (readIndex + tid) % BUFFER_SIZE : 0;

                    //get the pixel ID
                    unsigned int pixelID = stackData[readIndex];
                    //get the node ID
                    unsigned int nodeID = stackData[readIndex + BUFFER_SIZE];

                    //compute AABB intersection
                    bool intersect = rayBoxIntersect(nodes[nodeID].maxPoint.x, nodes[nodeID].maxPoint.y, nodes[nodeID].maxPoint.z,
                                                     nodes[nodeID].minPoint.x, nodes[nodeID].minPoint.y, nodes[nodeID].minPoint.z,
                                                     eyePos[0], eyePos[1], eyePos[2],
                                                     invDirections[pixelID][0], invDirections[pixelID][1], invDirections[pixelID][2]);

                    bool isLeaf = nodes[nodeID].isLeaf;
                    //is leaf
                    if(intersect && isLeaf){
                        //TODO : solve trig intersection...
                    }

                    //if the ray intersects, we add the children to the stack
                    int outputs = (intersect && !isLeaf) * 2;
                    int writeIndex = 0;

                    unsigned int leftChild = nodes[nodeID].left;
                    unsigned int rightChild = nodes[nodeID].right;

                    //reduce over all the outputs to get the write index and the total number of outputs
                    #pragma unroll
                    for(auto m = 0; m < tid; m++){
                        writeIndex += __shfl_sync(0xffffffff, outputs, m);
                    }

                    int writeIndexSrc = 0;

                    //cache write
                    //warp operation:
                    if((laneId == WARP_SIZE - 1 || laneId == queued - 1)){
                        writeIndexSrc = atomicAdd(stackPtrPending, writeIndex) - writeIndex;
                    }

                    if (intersect && !isLeaf){
                        //get write index
                        writeIndexSrc = __shfl_sync(0xffffffff, writeIndexSrc,
                                                    queued < WARP_SIZE ? queued - 1 : WARP_SIZE - 1);
                        //write the left child
                        stackData[writeIndexSrc + writeIndex - outputs] = pixelID;
                        stackData[writeIndexSrc + writeIndex - outputs + BUFFER_SIZE] = leftChild;
                        //write the right child
                        stackData[writeIndexSrc + writeIndex + 1 - outputs] = pixelID;
                        stackData[writeIndexSrc + writeIndex + 1 - outputs + BUFFER_SIZE] = rightChild;
                    }
                }
            }

            //more processing on the pixels ...


        }
    }

    void phongShading(material* materials, triangle* trigs, float3* vertices, AABBnode* nodes, pointLight* lights,
                      unsigned int numNodes, unsigned int numLights, CameraFrame* cameraFrame,float* imagePlane,
                      unsigned int numPixls, float3 ambientLight){
        unsigned int blockSize = 512;
        unsigned int gridSize = (numPixls/4 + blockSize - 1) / blockSize;
        auto t1 = std::chrono::high_resolution_clock::now();

        phongShadingDepthD < 512 ><<<gridSize, blockSize>>>(materials, trigs, vertices, nodes,
                lights, numNodes, numLights, cameraFrame, imagePlane,
                numPixls, ambientLight);

        hipDeviceSynchronize();
        auto t2 = std::chrono::high_resolution_clock::now();
        std::cout << "Time taken: " << std::chrono::duration_cast<std::chrono::milliseconds>(t2-t1).count() << "ms" << std::endl;
        assertCudaError();
    }
} // dylanrt